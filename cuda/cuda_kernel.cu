#include "hip/hip_runtime.h"
#define EIGEN_USE_GPU
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <fstream>
#include <assert.h>
#include <hip/hip_fp16.h>
#include <iostream>
#include <chrono>
#include <sys/time.h>
#include "error.cuh"
#include "gemm.cuh"
#include "reverseNswapdim23.cuh"
using namespace std;
#define THREADS_PER_BLOCK 1024
#define BLOCK_SIZE 1024
// #define MNT_MASK    0x7fffff
// #define EXP_MASK    0x7f800000
// #define SGN_MASK    0x80000000
// #define MAX_EXP     0xff
// #define BIAS        0x7f


#define T_SIZE 16
///************
#define MT 1

#define RD_NEAREST  0
#define RD_TOWZERO  1
#define RD_TOWPINF  2
#define RD_TOWNINF  3

__device__ unsigned long long int fn_MitchellMul_Optimized(unsigned int A, unsigned int B, unsigned int Sz)
{
	const unsigned int MNTSZ = 23;
	const unsigned int MNTSZ1 = 24;
	//-----------------------------------------------------------------
	unsigned long      int N = Sz;
	unsigned long long int one = 1;


	unsigned long long int charA = Sz-1;   // "8 -" is required because indexing in matlab is reverse than in the paper
	unsigned long long int charB = Sz-1;

	// May be different than depending on how I handle the fractional mantissa :: scaled or not scaled
	double mantA = (((double)A) / (one << charA) - 1);         // (10 Novemeber 2017) matissa is in fraction: I assume that the 52 bits in double precision are enough to retain the bits of the mantissa (which is integer in actual hardware implementation)
	double mantB = (((double)B) / (one << charB) - 1);


	// Get the log value in binary and in decimal respectively
	double lgAd = charA + mantA;
	double lgBd = charB + mantB;

	//--------------------------------------------------------------------------
	// Truncation
	unsigned long long int lgAv = (one << N) * (lgAd);  // (10 Novemeber 2017) Thefractional parts has given N Bits so that None of the bits from original number are lost
	unsigned long long int lgBv = (one << N) * (lgBd);  // (10 Novemeber 2017) Thefractional parts has given N Bits so that None of the bits from original number are lost

	//--------------------------------------------------------------------------

	// Adding the log of A and B
	unsigned long long int sumAB = lgAv + lgBv;
	// Converting it to binary
	double sumd = (double)sumAB / (one << N);

	// Extracting characteristic and mantissa part
	double charR = floor(sumd);
	double mantR = sumd - charR;

	double NormalizedAnswer = 1 + mantR;

	unsigned long long int Result;



	Result = (unsigned long long int) floor((NormalizedAnswer)* (one << (unsigned long long int)charR));


	if (A == 0 || B == 0)
		Result = 0;

	return Result;

}

__device__ unsigned long long int fn_mulN_Mitchell(unsigned int a, unsigned int b, unsigned int k)
{

	const unsigned int MNTSZ = 23;
	const unsigned int MNTSZ1 = 24;

//--------------------------------------------------------------------------
	unsigned long long int tA = MNTSZ;
	unsigned long long int tB = MNTSZ;

	unsigned long long int shamtA;
	unsigned long long int shamtB;

	unsigned long long int one = 1;

		shamtA = 24 - k;
		shamtB = 24 - k;

	unsigned long long int UA;
	unsigned long long int UB;
	unsigned long long int outL;

	UA = (a / (one << shamtA)) ;  // in integer form, UA plus the leading 1
	UB = (b / (one << shamtB)) ;

	outL = fn_MitchellMul_Optimized(UA , UB, k);

	return outL* (one << (shamtA  + shamtB ));

}




// Assumes inputs and outputs are in integer format
__device__ float FPMult_SinglePrecision_Rnone_Mitchell(float A, float B, unsigned long int k)
{
	unsigned long int round = RD_TOWZERO;
	// Reinterpretating the Value as an integer
	unsigned long int ai = *(unsigned int*)&A;
	unsigned long int bi = *(unsigned int*)&B;
	//------------------------------------------------------------------------------------------------------------------------
	// Given Parameters
	//------------------------------------------------------------------------------------------------------------------------
	const unsigned int EXPSZ = 8;
	const unsigned int MNTSZ = 23;
	//------------------------------------------------------------------------------------------------------------------------
	// Calcuated Parameters
	//------------------------------------------------------------------------------------------------------------------------
	const unsigned int P = MNTSZ + 1;   //% mantissa size with hidden bit

	const unsigned int MNT_MASK = (1 << MNTSZ) - 1;
	const unsigned int EXP_MASK = ((1 << EXPSZ) - 1) << MNTSZ;
	const unsigned int SGN_MASK = 1 << (EXPSZ + MNTSZ);

	const   signed int MAXEXP = (1 << EXPSZ) - 1;
	const   signed int BIAS = (1 << (EXPSZ - 1)) - 1;

	const		float REALMIN = pow(2, 1 - BIAS);

/*	//------------------------------------------------------------------------------------------------------------------------
	// Input Numbers and Accurate Output Calculation
	//------------------------------------------------------------------------------------------------------------------------
	float aF = -1e-7;
	float bF = 1-20;

	// logic for forcing subnormals to zero
	if (abs(aF) < REALMIN)
		aF = 0;
	if (abs(bF) < REALMIN)
		bF = 0;


	float oF = aF*bF;

	// Reinterpretating the Value as an integer
	unsigned long int ai = *(unsigned int*)&aF;
	unsigned long int bi = *(unsigned int*)&bF;


	*/
	//======================================================================================================================
	// FP Emulation STARTS HERE
	//======================================================================================================================


	// Separate the Sign bit, exponent and mantissa
	bool Asgn = (SGN_MASK & ai) != 0;
	bool Bsgn = (SGN_MASK & bi) != 0;

	signed long int Aexp = (EXP_MASK & ai) >> MNTSZ;
	signed long int Bexp = (EXP_MASK & bi) >> MNTSZ;

	unsigned long int Amnt = (MNT_MASK & ai);		// The hidden bit is not appended yet. In MATLAB or Verilog code, the hidden bit is appended at this point
	unsigned long int Bmnt = (MNT_MASK & bi);

	//--------------------------------------------------------------------------

	//----Exceptions Handling----
	// Flags for input and output flag decisions based on input
	bool	Azero = (Aexp == 0);
	bool	Bzero = (Bexp == 0);
	bool	Inzero = Azero | Bzero;

	bool	Ainf = (Aexp == MAXEXP);
	bool	Binf = (Bexp == MAXEXP);
	bool	Ininf = Ainf | Binf;          // will be high for both Nan or Inf

	// excp and checking value of msb of mantissa(if that is 1, it is a Nan otherwise Inf)
	bool	Anan = Ainf & (Amnt  != 0);      // Checking the MSB of the mantissa. If it is one, then its a NaN. I could also check here if Amnt> (2^(MNT-1))
	bool	Bnan = Binf & (Bmnt  != 0);

	bool	Innan = Anan | Bnan;

	//------------------------------
	// Initializing Output Flags
	bool	flag_zero0 = 0, flag_zero1 = 0, flag_zero3 = 0;
	bool	flag_nan0 = 0;
	bool	flag_inf0 = 0, flag_inf1 = 0, flag_inf2 = 0, flag_inf3 = 0;


	flag_zero0 = Inzero & !Ininf;            // it may be high even if the output should be nan.so nan flag should be given most prioriy
	flag_nan0 = (Inzero &  Ininf) || Innan;  // The " OR Innan" part will be covered in Ininf because;
	flag_inf0 = !Inzero &  Ininf;            // it may be high even if the output should be nan.so nan flag should be given most prioriy


	//--------------------------------------------------------------------------
	// Sign Calculation
	//--------------------------------------------------------------------------
	bool Osgn = Asgn^Bsgn;

	//--------------------------------------------------------------------------
	// Exponent Calculation
	// --------------------------------------------------------------------------
	signed long int	Oexp1 = (Aexp + Bexp - BIAS);

	flag_zero1 = (Oexp1 < 0);
	flag_inf1 = (Oexp1 >= MAXEXP);


	//**************************************************************************
	// Mantissa Calculation (Main computation of the Code)
	//**************************************************************************
	unsigned long long int	Mult = fn_mulN_Mitchell(((1 << MNTSZ) + Amnt), ((1 << MNTSZ) + Bmnt), k);  // not sure if it will get the right number of bits or it will get truncation
	unsigned long long int	Normd;
	signed	      long int	Oexp2;
	//-------------------------------------------
	//normalization after mantissa multiplication
	//-------------------------------------------
	if (Mult >> (2 * P - 1))
	{
		Normd = Mult;
		Oexp2 = Oexp1 + 1; //%% --IMPORTANT -- NEED to check Exception here %
	}
	else
	{
		Normd = Mult << 1;
		Oexp2 = Oexp1;
	}


	flag_inf2 = (Oexp2 == MAXEXP);

	//------------------------------
	// Rounding (NO rounding)
	// -----------------------------
	//rb = 0;
	//unsigned long long int Rounded = (Normd >> P) + rb;		// Truncate 24 lower bits from the multiplier result and add rounding bit
	unsigned long long int Rounded = (Normd >> P);		// Truncate 24 lower bits from the multiplier result and add rounding bit

	//-------------------------------------------
	// normalization after rounding (NOT NEEDED)
	//-------------------------------------------
	unsigned long long int Rounded2;
	signed	      long int	Oexp3;
	// Check if there is carry from rounding and adjust
	/*if (Rounded >> P)						// check if 25th bit is one
	{
	Rounded2 = Rounded >> 1;
	Oexp3 = Oexp2 + 1;					//--IMPORTANT -- NEED to check Exception here %
	}
	else
	{*/
	Rounded2 = Rounded;
	Oexp3 = Oexp2;
	//}

	flag_inf3 = (Oexp3 == MAXEXP);
	flag_zero3 = (Oexp3 == 0);
	//**************************************************************************
	//  Putting together the Output
	//**************************************************************************
	bool	flag_zero = flag_zero0 | flag_zero1 | flag_zero3;
	bool	flag_nan = flag_nan0;
	bool	flag_inf = flag_inf0 | flag_inf1 | flag_inf2 | flag_inf3;

	unsigned long int Omnt;
	signed long int Oexp4;

	unsigned int ResultCase = 4 * (flag_nan)+2 * (flag_inf)+(flag_zero);

	switch (ResultCase)
	{
	case 0:					// Normal case
		Omnt = Rounded2;
		Oexp4 = Oexp3;
		break;
	case 1:					// Zero
		Oexp4 = 0;
		Omnt = 0;
		break;
	case 2:					// Infinity
		Oexp4 = MAXEXP;
		Omnt = 0;
		break;
	case 4:					// NaN
		Oexp4 = MAXEXP;
		Omnt = 1 << (MNTSZ - 1);
		break;
	default:				// For every other case, set output as Nan(it depend on our choice how we want to handle the error)
		Oexp4 = MAXEXP;
		Omnt = 1 << (MNTSZ - 1);
		break;
	}


	unsigned long int Os = (((unsigned int)(Osgn & ~flag_nan)) << (EXPSZ + MNTSZ));
	unsigned long int Oe = (Oexp4 << MNTSZ);
	unsigned long int Om = Omnt & MNT_MASK;

	unsigned long int Oi = Os + Oe + Om;


	float O = *(float*)&Oi;


	return O;
}
///***************

static inline double realtime(void) {
    struct timeval tp;
    struct timezone tzp;
    gettimeofday(&tp, &tzp);
    return tp.tv_sec + tp.tv_usec * 1e-6;
}

__device__ float halfmul(const float a, const float b){


      half A = __float2half(a);
      half B = __float2half(b);
      half C;
  #if __CUDA_ARCH__ >= 530
      C = __hmul(A, B);
  #else
      C = __float2half(__half2float(A)*__half2float(B));
  #endif
      float c = __half2float(C);

    return c;
}

__device__ float bitmasking(float num){
	int mask = 0xffff0000;
	//convert to int
	int b = *(int*)&num;
    int masked = b&mask;
    float ret  = *(float*)&masked;
	return ret;
}

__device__ unsigned long long  int fn_MitchellMul_Optimized_Unbiased_LowerBitsReduced_copy(unsigned int A, unsigned int B, unsigned int Sz)
{
	const unsigned int MNTSZ = 23;
	const unsigned int MNTSZ1 = 24;
	//-----------------------------------------------------------------
	unsigned long      int N = Sz;
	unsigned long long int one = 1;


	unsigned long long int charA = Sz-1;   // "8 -" is required because indexing in matlab is reverse than in the paper
	unsigned long long int charB = Sz-1;

	// May be different than depending on how I handle the fractional mantissa :: scaled or not scaled
	double mantA = (((double)A) / (one << charA) - 1);         // (10 Novemeber 2017) matissa is in fraction: I assume that the 52 bits in double precision are enough to retain the bits of the mantissa (which is integer in actual hardware implementation)
	double mantB = (((double)B) / (one << charB) - 1);


	// Get the log value in binary and in decimal respectively
	double lgAd = charA + mantA;
	double lgBd = charB + mantB;

	//--------------------------------------------------------------------------
	// Truncation
	unsigned long long int lgAv = (one << N) * (lgAd);  // (10 Novemeber 2017) Thefractional parts has given N Bits so that None of the bits from original number are lost
	unsigned long long int lgBv = (one << N) * (lgBd);  // (10 Novemeber 2017) Thefractional parts has given N Bits so that None of the bits from original number are lost


	//--------------------------------------------------------------------------

	// Adding the log of A and B
	unsigned  long long int sumAB = lgAv + lgBv;

	// Converting it to binary
	double sumd = (double)sumAB / (one << N);

	// Extracting characteristic and mantissa part
	unsigned  long long int charR = floor(sumd);
	double mantR = sumd - charR;



	double CorrTem = (1.0 / 16 + 1.0 / 64) / (one << (charR - (charA + charB))); //2^-4+2^-6; // ==0.0781;
	//------------------------
	//Edit for lower bit reduction
	unsigned int chopbits = (Sz - 1);
	CorrTem = floor((one << chopbits)*(CorrTem)) / (one << chopbits);
	//----------------------

	double NormalizedAnswer = 1.0 + mantR + CorrTem;  // Here, since the answers are in fraction, the Corr Terms precision wont be lost. (Which is important to keep the peak error low).
	// However, when charr is lessthan or equal to 6, these bits are lost in the
	// final shifting. Therefore a corner case needs to be added

	//-----------------------------------------------------------------
	if ((charR == (2 * N - 1)) && (NormalizedAnswer >= 2))
		// cornercase: overflow
		NormalizedAnswer = 1 + mantR;
	//-----------------------------------------------------------------


	unsigned  long long  int Result;

	Result = (unsigned long long  int)floor((NormalizedAnswer)*(one << charR));

	//-----------------------------------------------------------------
	if ((charR <= 6) && (charR - (charA + charB) == 1) && Sz >= 8)
		// cornercase: peak error
		Result = Result + 1;
	//-----------------------------------------------------------------

	if (A == 0 || B == 0)
		Result = 0;


	return Result;

}



__device__ unsigned long long int fn_mulN_UREMrd(unsigned int a, unsigned int b, unsigned int k)
{

	const unsigned int MNTSZ = 23;
	const unsigned int MNTSZ1 = 24;

//--------------------------------------------------------------------------
	unsigned long long int tA = MNTSZ;
	unsigned long long int tB = MNTSZ;

	unsigned long long int shamtA;
	unsigned long long int shamtB;

	unsigned long int lA = 0;
	unsigned long int lB = 0;
	unsigned long long int one = 1;

	if (MNTSZ1 > k)
	{
		shamtA = MNTSZ - k + 2;
		lA = 1;
	}
	else
		shamtA = 0;

	if (MNTSZ1 > k)
	{
		shamtB = MNTSZ - k + 2;
		lB = 1;
	}
	else
		shamtB = (0);


	unsigned long long int UA;
	unsigned long long int UB;
	unsigned long long int outL;

	UA = ((a >> shamtA)  << lA) + lA;  // in integer form, UA plus the leading 1
	UB = ((b >> shamtB)  << lB) + lB;

	outL = fn_MitchellMul_Optimized_Unbiased_LowerBitsReduced_copy(UA,UB,k);

	return (outL  << ((shamtA - lA) + (shamtB - lB)) );

}




/* The gateway function */
__device__ float FPmultMBM_cppv2(float Af, float Bf, int t)
{
	//=============================================================================
    // 1ST FUNCTION BODY
    //=============================================================================
    unsigned long int ai = *(unsigned int*)&Af;
	unsigned long int bi = *(unsigned int*)&Bf;

	//unsigned long int bitmask =  ~((1 << t) - 1);   //==> 2^t-1;

	unsigned long int at = ai; // &bitmask; let the integer multiplier handle the truncation. In old work, I was truncating both input and output. but not now. So this is not needed
	unsigned long int bt = bi; // &bitmask;

	//========================================== MITCHELLs MULTIPLICATION FUNCTION (EQUIVALENT TO MATLAB IS HERE)======================================

	//------------------------------------------------------------------------------------------------------------------------
	// Given Parameters
	//------------------------------------------------------------------------------------------------------------------------
	const unsigned int EXPSZ = 8;
	const unsigned int MNTSZ = 23;
	//------------------------------------------------------------------------------------------------------------------------
	// Calcuated Parameters
	//------------------------------------------------------------------------------------------------------------------------
	const unsigned int P = MNTSZ + 1;   //% mantissa size with hidden bit
	int k = P - t;				// The input parameter t is comming as the number of bits truncated.
								// my new implementation of DRUM and MBM takes k as the parameter which means the number of bits retained


	const unsigned int MNT_POS = (1 << MNTSZ);
	const unsigned int MNT_MASK = MNT_POS - 1;

	const unsigned int EXP_MASK = ((1 << EXPSZ) - 1) << MNTSZ;
	const unsigned int SGN_MASK = 1 << (EXPSZ + MNTSZ);

	const   signed int MAXEXP = (1 << EXPSZ) - 1;
	const   signed int BIAS = (1 << (EXPSZ - 1)) - 1;

	//const	float REALMIN = pow(2, 1 - BIAS);

	////********************************************************************************************************************************************
	//======================================================================================================================
	// FP Emulation STARTS HERE
	//======================================================================================================================
	// Separate the Sign bit, exponent and mantissa
	bool Asgn = (SGN_MASK & at) != 0;
	bool Bsgn = (SGN_MASK & bt) != 0;

	signed long int Aexp = (EXP_MASK & at) >> MNTSZ;
	signed long int Bexp = (EXP_MASK & bt) >> MNTSZ;

	unsigned long int Amnt = (MNT_MASK & at);		// The hidden bit is not appended yet. In MATLAB or Verilog code, the hidden bit is appended at this point
	unsigned long int Bmnt = (MNT_MASK & bt);

	//--------------------------------------------------------------------------

	//----Exceptions Handling----
	// Flags for input and output flag decisions based on input
	bool	Azero = (Aexp == 0);
	bool	Bzero = (Bexp == 0);
	bool	Inzero = Azero | Bzero;

	bool	Ainf = (Aexp == MAXEXP);
	bool	Binf = (Bexp == MAXEXP);
	bool	Ininf = Ainf | Binf;          // will be high for both Nan or Inf

	// excp and checking value of msb of mantissa(if that is 1, it is a Nan otherwise Inf)
	bool	Anan = Ainf & (Amnt  != 0);      // Checking the MSB of the mantissa. If it is one, then its a NaN. I could also check here if Amnt> (2^(MNT-1))
	bool	Bnan = Binf & (Bmnt  != 0);

	bool	Innan = Anan | Bnan;

	//------------------------------
	// Initializing Output Flags
	bool	flag_zero0 = 0, flag_zero1 = 0, flag_zero3 = 0;
	bool	flag_nan0 = 0;
	bool	flag_inf0 = 0, flag_inf1 = 0, flag_inf2 = 0, flag_inf3 = 0;


	flag_zero0 = Inzero & !Ininf;            // it may be high even if the output should be nan.so nan flag should be given most prioriy
	flag_nan0 = (Inzero &  Ininf) || Innan;  // The " OR Innan" part will be covered in Ininf because;
	flag_inf0 = !Inzero &  Ininf;            // it may be high even if the output should be nan.so nan flag should be given most prioriy


	//--------------------------------------------------------------------------
	// Sign Calculation
	//--------------------------------------------------------------------------
	bool Osgn = Asgn^Bsgn;

	//--------------------------------------------------------------------------
	// Exponent Calculation
	// --------------------------------------------------------------------------
	signed long int	Oexp1 = (Aexp + Bexp - BIAS);

	flag_zero1 = (Oexp1 < 0);
	flag_inf1 = (Oexp1 >= MAXEXP);


	//**************************************************************************
	// Mantissa Calculation (Main computation of the Code)
	//**************************************************************************
	unsigned long long int	Mult = fn_mulN_UREMrd(((1 << MNTSZ) + Amnt), ((1 << MNTSZ) + Bmnt), k);  // not sure if it will get the right number of bits or it will get truncation
	//	unsigned long long int	Mult = MitchelOptimIntMult(Amnt, Bmnt);  // Passing without the hidden bit as it is always known

	//#############################################################################################################################

	unsigned long long int	Normd;
	signed	      long int	Oexp2;
	//-------------------------------------------
	//normalization after mantissa multiplication
	//-------------------------------------------
	if (Mult >> (2 * P - 1))
	{
		Normd = Mult;
		Oexp2 = Oexp1 + 1; //%% --IMPORTANT -- NEED to check Exception here %
	}
	else
	{
		Normd = Mult << 1;
		Oexp2 = Oexp1;
	}

	flag_inf2 = (Oexp2 == MAXEXP);

	//------------------------------
	// Rounding (NO rounding)
	// -----------------------------
	//rb = 0;
	//unsigned long long int Rounded = (Normd >> P) + rb;		// Truncate 24 lower bits from the multiplier result and add rounding bit
	unsigned long long int Rounded = (Normd >> P);		// Truncate 24 lower bits from the multiplier result and add rounding bit

	//-------------------------------------------
	// normalization after rounding (NOT NEEDED)
	//-------------------------------------------
	unsigned long long int Rounded2;
	signed	      long int	Oexp3;
	// Check if there is carry from rounding and adjust
	/*if (Rounded >> P)						// check if 25th bit is one
	{
	Rounded2 = Rounded >> 1;
	Oexp3 = Oexp2 + 1;					//--IMPORTANT -- NEED to check Exception here %
	}
	else
	{*/
	Rounded2 = Rounded;
	Oexp3 = Oexp2;
	//}

	flag_inf3 = (Oexp3 == MAXEXP);
	flag_zero3 = (Oexp3 == 0);

	//**************************************************************************
	//  Putting together the Output
	//**************************************************************************
	bool	flag_zero = flag_zero0 | flag_zero1 | flag_zero3;
	bool	flag_nan = flag_nan0;
	bool	flag_inf = flag_inf0 | flag_inf1 | flag_inf2 | flag_inf3;

	unsigned long int Omnt;
	signed long int Oexp4;

	unsigned int ResultCase = 4 * (flag_nan)+2 * (flag_inf)+(flag_zero);

	switch (ResultCase)
	{
	case 0:					// Normal case
		Omnt = Rounded2;
		Oexp4 = Oexp3;
		break;
	case 1:					// Zero
		Oexp4 = 0;
		Omnt = 0;
		break;
	case 2:					// Infinity
		Oexp4 = MAXEXP;
		Omnt = 0;
		break;
	case 4:					// NaN
		Oexp4 = MAXEXP;
		Omnt = 1 << (MNTSZ - 1);
		break;
	default:				// For every other case, set output as Nan(it depend on our choice how we want to handle the error)
		Oexp4 = MAXEXP;
		Omnt = 1 << (MNTSZ - 1);
		break;
	}


	unsigned long int Os = (((unsigned int)(Osgn & ~flag_nan)) << (EXPSZ + MNTSZ));
	unsigned long int Oe = (Oexp4 << MNTSZ);
	unsigned long int Om = Omnt & MNT_MASK;

	unsigned long int Oi = Os + Oe + Om;

    //unsigned long int Oi = MitchelFPMultiply(at, bt);

    /////*******************************************************************************************************************************************


	//==============================================************ ENDS HERE ********=================================================

	// Truncating Output Bits
	//	unsigned long int Oi = *(unsigned int*)&Of;
	unsigned long int Ot = Oi; //& bitmask;
	float Oft = *(float*)&Ot;
    //=============================================================================

	return Oft;

}
//=============================================================================
//===============================IM2COL KERNEL=================================
//=============================================================================
/*po patch offset, pc patch count*/
__global__ void im2col_improved(const float *in,
    int c, int w, int h, int ow, int oh,
    int kw, int kh, int pw, int ph, int sw, int sh,
    int dw, int dh, int po, int pc, float *out)
{
//pc = ow * oh * batch aka m dimension
unsigned pl = kw * kh * c;
for(unsigned tId = blockIdx.x * blockDim.x + threadIdx.x; tId < pc*pl; tId += blockDim.x * gridDim.x)
{
    unsigned patchId = (tId + po*pl) / pl;
    unsigned outB    = (patchId / ow) / oh;
    unsigned outH    = (patchId / ow) % oh;
    unsigned outW    = patchId % ow;

    unsigned valueId = (tId + po*pl) % pl;
    unsigned offsetH = valueId / (kw * c);
    unsigned offsetW = (valueId / c) % kw;
    unsigned offsetC = valueId % c;

    int inH = outH * sh - ph + offsetH * dh;
    int inW = outW * sw - pw + offsetW * dw;

    if(inH >= 0 && inW >= 0 && inH < h && inW < w)
        out[tId] = in[((outB * h + inH) * w + inW) * c + offsetC];
    else
        out[tId] = float(0);

}

}
//=============================================================================
//=============================================================================
//=============================================================================
void im2colLauncher_Improved(
    const float* im,
    const int batch,
    const int in_row,
    const int in_col,
    const int out_row,
    const int out_col,
    const int out_depth,
    const int in_depth,
    const int filter_row,
    const int filter_col,
    const int stride_row,
    const int stride_col,
    // Padding
    const int left_offset,
    const int top_offset,
    const int dw,
    const int dh,
    float* data_col)
{

    unsigned pl = filter_row * filter_col * in_depth;
    unsigned blockSize = 256;
    unsigned gridSize  = (batch * pl + blockSize - 1) / blockSize;
    im2col_improved<<<gridSize,blockSize,0>>>(im, in_depth, in_col, in_row, out_col, out_row, filter_col, filter_row,  left_offset,top_offset, stride_col, stride_row,dw,dh,0,batch*out_row*out_col,data_col);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

}

// //=============================================================================
// //===============================IM2COL KERNEL=================================

__global__ void im2col_improved_filtergrad(const float *in, int batch,
    int c, int w, int h, int ow, int oh,
    int kw, int kh, int pw, int ph, int sw, int sh,
    int dw, int dh, int po, int pc, float *out)
{
//unsigned pc = kernel_height * kernel_width * channel_in

unsigned pl = batch * oh * ow;
for(unsigned tId = blockIdx.x * blockDim.x + threadIdx.x; tId < pc*pl; tId += blockDim.x * gridDim.x)
{
    unsigned patchId = (tId + po*pl) / pl;
    unsigned outB    = (patchId / c) / kw; // kh
    unsigned outH    = (patchId / c) % kw; // kw
    unsigned outW    = patchId % c; // c

    unsigned valueId = (tId + po*pl) % pl; // element position in window
    unsigned offsetH = valueId / (ow * oh);//ob
    unsigned offsetW = (valueId / ow) % oh;//oh
    unsigned offsetC = valueId % ow; //ow

    int inH = outB * 1 - ph + offsetW * 1;
    int inW = outH * 1 - pw + offsetC * 1;
    if(inH >= 0 && inW >= 0 && inH < h && inW < w)
        out[tId] = in[((offsetH * h + inH) * w + inW) * c + outW];
    else
        out[tId] = float(0);

}

}
//=============================================================================
//=============================================================================
//=============================================================================
void im2colLauncher_Improved_filtergrad(
    const float* im,
    const int batch,
    const int in_row,
    const int in_col,
    const int out_row,
    const int out_col,
    const int out_depth,
    const int in_depth,
    const int filter_row,
    const int filter_col,
    const int stride_row,
    const int stride_col,
    // Padding
    const int left_offset,
    const int top_offset,
    const int dw,
    const int dh,
    float* data_col)
{
    unsigned pl = batch * out_row * out_col;
    unsigned blockSize = 256;
    unsigned gridSize  = (filter_row * pl + blockSize - 1) / blockSize;
    // __global__ void im2col_improved_filtergrad(const float *in, int batch,
    // int c, int w, int h, int ow, int oh,
    // int kw, int kh, int pw, int ph, int sw, int sh,
    // int dw, int dh, int po, int pc, float *out)
    im2col_improved_filtergrad<<<gridSize,blockSize,0>>>(im, batch, in_depth, in_col, in_row, out_col, out_row, filter_col, filter_row,  left_offset,top_offset, stride_col, stride_row,dw,dh,0,filter_row*filter_col*in_depth,data_col);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

}

void gemm_reference( size_t m, size_t n,
    size_t k, const float* a, size_t lda, const float* b, size_t ldb,
    float* c, size_t ldc){
    const size_t a_i_stride = lda; 
    const size_t b_l_stride = ldb; 
    const size_t c_i_stride = ldc; 
    size_t i, j, l;
    // loop output depth
    for (j = 0; j < n; j++) {
      //loop patch
      for (i = 0; i < m; i++) {
        float total(0);
        //loop filter_value_count
        for (l = 0; l < k; l++) {
          const size_t a_index = ((i * a_i_stride) + l );
          const float a_value = a[a_index];
          // filte
          const size_t b_index = (j  + (l * b_l_stride));
          const float b_value = b[b_index];
          total += (a_value * b_value);
        }
        const size_t c_index = ((i * c_i_stride) + j );
        c[c_index] = total;
      }
    }
  }


// HWC -> (C*f_h*f_w)*(out_row*out_col)
__global__ void im2col(
    const int size,
    const float* im,
    const int in_row,
    const int in_col,
    const int filter_row,
    const int filter_col,
    const int left_offset,
    const int top_offest,
    const int stride_row,
    const int stride_col,
    const int in_channel,
    //output height width
    const int height_cols,
    const int width_cols,
    int im_stride,
    int vec_stride,
    float* data_vec)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // let's narrow down our sight into one batch (each batch will handle by blockIdx.y)
    // size is equal to output vectorized amount i.e. HWC (height * witdh * channel)
    if (index < size){
        const int batch_idx = blockIdx.y;
        // select current image batch HWC format
        im += batch_idx * im_stride;
        // select current vectorized batch (f_h * f_w * channel) * (output_row * output_col)
        data_vec += batch_idx * vec_stride;

        const int h_index = index / in_channel;
        // 1
        const int c_im = index % in_channel;
        // 0
        const int w_col = h_index % width_cols;
        // 0
        const int h_col = h_index / width_cols;
        // 1
        const int c_col = c_im;
        // 0
        const int h_offset = h_col * stride_row - top_offest;
        const int w_offset = w_col * stride_col - left_offset;


        //index = c + indepth*COL + indepth*col*ROW
        const float* im_ptr = im;
        // 1
        im_ptr +=(h_offset * in_col + w_offset) * in_channel + c_im;
        float* vec_ptr = data_vec;
        //HWC
        vec_ptr += (h_col * width_cols+ w_col) * in_channel * filter_col * filter_row + c_col;

        for(int i = 0; i < filter_row; i++){
            for(int j = 0; j < filter_col; j++){
                int h_im = h_offset + i;
                int w_im = w_offset + j;
                // *vec_ptr = (h_im >= 0 && w_im >= 0 && h_im < in_row && w_im < in_col )? batch_idx * im_stride+(h_offset * in_col + w_offset) * in_channel + c_im+(i*in_col+j)*in_channel:0;
                *vec_ptr = (h_im >= 0 && w_im >= 0 && h_im < in_row && w_im < in_col )? im_ptr[(i*in_col+j)*in_channel]:0;

                vec_ptr += in_channel;

            }
        }

    }

}




void im2colLauncher(
    const float* im,
    const int batch,
    const int in_row,
    const int in_col,
    const int out_row,
    const int out_col,
    const int out_depth,
    const int in_depth,
    const int filter_row,
    const int filter_col,
    const int stride_row,
    const int stride_col,
    // Padding
    const int left_offset,
    const int top_offset,
    float* data_col)
{
    int height_col = out_row;
    int witdh_col = out_col;
    int size = in_depth * height_col * witdh_col;
    // number of elements in one batch of input
    int im_stride = in_depth * in_row * in_col;
    // number of element in one batch of vectorized output
    int vec_stride = in_depth * filter_row * filter_col * out_row * out_col;
   // printf("size %d, im_stride %d, vec_stride %d, %d leftoffset, %d topoffset\n",size, im_stride, vec_stride,left_offset,top_offset);
    dim3 dim_grid(ceil((float)size/BLOCK_SIZE),batch);

    im2col<<<dim_grid,BLOCK_SIZE>>>(
        size, im, in_row, in_col, filter_row, filter_col, left_offset, top_offset,
        stride_row, stride_col, in_depth, height_col, witdh_col, im_stride, vec_stride, data_col
    );

    // printf("size %d, in_row %d, in_col %d, filter_row %d, filter_col %d, left_offset %d, top_offset %d, stride_row %d, stride_col %d, in_depth %d, height_col %d, width_col %d\n",size,in_row, in_col, filter_row, filter_col, left_offset, top_offset,
    // stride_row, stride_col, in_depth, height_col, witdh_col);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

}


void ConvamKernellLauncher(
    const float* inputs,
    const float* filter,
    float* im2col,
    const int batch,
    const int in_row,
    const int in_col,
    const int out_row,
    const int out_col,
    const int out_depth,
    const int in_depth,
    const int filter_row,
    const int filter_col,
    const int stride_row,
    const int stride_col,
    // Padding
    const int left_offset,
    const int top_offset,
    const int padding,
    float* output
  ){

    
    if (filter_row == 1 && filter_col == 1 && stride_row == 1 &&
        stride_col == 1) {
      // The kernel is 1x1.
      const int m = batch * in_row * in_col;
      const int n = out_depth;
      const int k = in_depth;
      const int lda = k;
      const int ldb = n;
      const int ldc = n;
      const int size = m*n;
      dim3 blockSize(16, 16, 1);
      dim3 gridSize((n + blockSize.x - 1) / blockSize.x, (m + blockSize.y - 1) / blockSize.y, 1);
      gemm<<<gridSize,blockSize,0>>>(m,n,k,inputs,lda,filter,ldb,output,ldc);
      gpuErrchk( hipPeekAtLastError() );
gpuErrchk( hipDeviceSynchronize() );
      return;
    } else if (filter_row == in_row && filter_col== in_col &&
               padding == 1) {
      // The input data and filter have the same height/width.
      const int m = batch;
      const int n = out_depth;
      const int k = in_depth*in_col*in_row;
      const int lda = k;
      const int ldb = out_depth;
      const int ldc = out_depth;
      const int size = m*n;
      dim3 blockSize(16, 16, 1);
      dim3 gridSize((n + blockSize.x - 1) / blockSize.x, (m + blockSize.y - 1) / blockSize.y, 1);
      gemm<<<gridSize,blockSize,0>>>(m,n,k,inputs,lda,filter,ldb,output,ldc);
      gpuErrchk( hipPeekAtLastError() );
gpuErrchk( hipDeviceSynchronize() );
      return;
    }
    double begin = realtime();
   im2colLauncher_Improved(inputs, batch, in_row, in_col, out_row, out_col,out_depth, in_depth, filter_row, filter_col, stride_row, stride_col, left_offset,top_offset, 1,1 ,im2col);
   hipDeviceSynchronize();
   double end = realtime();
#ifdef PROFILE
cout << "Forward Im2col time difference = " << end - begin << endl;
#endif
   const size_t m = batch*out_row*out_col; 
   const size_t n = out_depth; 
   const size_t k = filter_col * filter_row * in_depth; 
   const size_t lda = k; 
   const size_t ldb = out_depth;
   const size_t ldc = out_depth;
   dim3 blockSize(16, 16, 1);
   dim3 gridSize((n + blockSize.x - 1) / blockSize.x, (m + blockSize.y - 1) / blockSize.y, 1);
    begin =realtime();
   gemm<<<gridSize,blockSize,0>>>(m,n,k,im2col,lda,filter,ldb,output,ldc);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    end = realtime();
#ifdef PROFILE
    cout << "Forward gemm time difference = " << end - begin << endl;
#endif
    // const int64 kMaxChunkSize = (16 * 1024 * 1024) / sizeof(float);
    // int64 patchLength  = filter_col*filter_row*in_depth;
    // int64 totalPatchesCount = batch * out_row * out_col;
    // const int64 patchesPerChunk = kMaxChunkSize / patchLength;
    // for(int64 i = 0; i < totalPatchesCount; i += patchesPerChunk)
    // {
    //     int64 temp_batch = i/(out_row*out_col);
    //     int patchOffset = int(i % (out_row * out_col));
    //     int patchesCount = int(min(patchesPerChunk, totalPatchesCount - i));
    //     const float *patchInputData =
    // }


  }

void Im2col(const float* input_data, const int depth, const int height,
    const int width, const int filter_h, const int filter_w,
    const int pad_t, const int pad_l, const int pad_b, const int pad_r,
    const int stride_h, const int stride_w, float* col_data) {
    int height_col = 3;
    int width_col = 3;

    int h_pad = -pad_t;
    for (int h = 0; h < height_col; ++h) {
        int w_pad = -pad_l;
            for (int w = 0; w < width_col; ++w) {
                for (int ih = h_pad; ih < h_pad + filter_h; ++ih) {
                    for (int iw = w_pad; iw < w_pad + filter_w; ++iw) {
                    if (ih >= 0 && ih < height && iw >= 0 && iw < width) {
                        memcpy(col_data, input_data + (ih * width + iw) * depth,
                            sizeof(float) * depth);
                    } else {
                        // This should be simply padded with zero.
                        memset(col_data, 0, sizeof(float) * depth);
                    }
                col_data += depth;
                }
            }
        w_pad += stride_w;
        }
        h_pad += stride_h;
    }
}


__global__ void filtergradkernel(
    const int size,
    const int OUT_CHANNEL,
    const int IN_CHANNEL,
    const int FILTER_COL,
    const int FILTER_ROW,
    const int BATCH,
    const int HOLE_GRAD_HEIGHT,
    const int HOLE_GRAD_WIDTH,
    const int GRAD_HEIGHT,
    const int GRAD_WIDTH,
    const int STRIDE_ROW,
    const int STRIDE_COL,
    const int INPUT_HEIGHT,
    const int INPUT_WIDTH,
    const int LEFT_OFFSET,
    const int TOP_OFFSET,
    const float* input,
    const float* grad,
    float* out
){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size){
        // index = out_channel + OUT_CHANNEL*(in_channel + IN_CHANNEL*(col + FILTER_COL*row))
        const int out_channel = index % OUT_CHANNEL;
        // idx_filter = in_channel + IN_CHANNEL*(col + FILTER_COL*row)
        const int idx_filter = index / OUT_CHANNEL;
        const int in_channel = idx_filter % IN_CHANNEL;
        // idx_filter_slice = col + FILTER_COL*row
        const int idx_filter_slice = idx_filter / IN_CHANNEL;
        const int col = idx_filter_slice % FILTER_COL;
        const int row = idx_filter_slice / FILTER_COL;

        float temp = 0;
        for(int i = 0; i < BATCH; ++i){
            for(int j = 0; j < HOLE_GRAD_HEIGHT; ++j){
                for(int k = 0; k < HOLE_GRAD_WIDTH; ++k){

                    const float i_row = j/(float)STRIDE_ROW;
                    const float i_col = k/(float)STRIDE_COL;
                    const bool y = fmod(i_row,(float)1)==float(0);
                    const bool x = fmod(i_col,(float)1)==float(0);
                    float grad_val = (x&y)?grad[i*GRAD_WIDTH*GRAD_HEIGHT*OUT_CHANNEL + int(i_row)*GRAD_WIDTH*OUT_CHANNEL+ int(i_col)*OUT_CHANNEL + out_channel]:0;
                    // float input_val = input[i*INPUT_HEIGHT*INPUT_WIDTH*IN_CHANNEL++ in_channel];
                    const int input_row = row - TOP_OFFSET + j;
                    const int input_col = col - LEFT_OFFSET + k;
                    float input_val = 0;
                    if( input_row >= 0 && input_col >=0 && input_row < INPUT_HEIGHT && input_col < INPUT_WIDTH){
                        input_val = input[i*INPUT_WIDTH*INPUT_HEIGHT*IN_CHANNEL+ input_row*INPUT_WIDTH*IN_CHANNEL + input_col*IN_CHANNEL+ in_channel];
                    }
                   // temp += FPMult_SinglePrecision_Rnone_Mitchell(input_val,grad_val,MT);
                    //temp += FPmultMBM_cppv2(input_val , grad_val,T_SIZE);
                    //bf16*bf16 -> 1+8+14 = 23bits
                    // temp += bitmasking(bitmasking(input_val)*bitmasking(grad_val));
                    //temp+= halfmul(input_val,grad_val);
temp += input_val*grad_val;
                }
            }
        }
        out[index] = temp;
    }
}
void ConvamFilterGradKernelLauncher(
    const float* input,
    const float* grad,
    float* im2col,
    const int input_height,
    const int input_width,
    const int batch,
    const int in_depth,
    const int grad_width,
    const int grad_height,
    const int grad_channel,
    const int filter_left_offset,
    const int filter_top_offset,
    const int stride_row,
    const int stride_col,
    const int filter_width,
    const int filter_height,
    float* out
){

    // im2colLauncher_Improved(input,filter_height,)
    // const int total_size = filter_height*filter_width*in_depth*grad_channel;
    // const int grid_size = (total_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    // double begin = realtime();
    // filtergradkernel<<<grid_size,BLOCK_SIZE>>>( total_size, grad_channel, in_depth, filter_width, filter_height, batch, ((grad_height-1)*stride_row+1), ((grad_width-1)*stride_col+1),
    // grad_height, grad_width, stride_row, stride_col, input_height, input_width, filter_left_offset, filter_top_offset, input, grad, out

    // );
    
    // gpuErrchk( hipPeekAtLastError() );
    // gpuErrchk( hipDeviceSynchronize() );
    // double end = realtime();
    double begin = realtime();
    im2colLauncher_Improved_filtergrad(input,batch,input_height,input_width,grad_height,grad_width,grad_channel,in_depth,filter_height,filter_width,1,1,\
    filter_left_offset,filter_top_offset,1,1,im2col);
    double end = realtime();
#ifdef PROFILE
    cout << "Filter gradient im2col difference = " << end - begin << endl;
#endif
    const size_t m = filter_height*filter_width*in_depth; 
    const size_t n = grad_channel; 
    const size_t k = batch*grad_height*grad_width; 
    const size_t lda = k; 
    const size_t ldb = n;
    const size_t ldc = n;

    begin =realtime();
    dim3 blockSize(16, 16, 1);
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x, (m + blockSize.y - 1) / blockSize.y, 1);
    gemm<<<gridSize,blockSize,0>>>(m,n,k,im2col,lda,grad,ldb,out,ldc);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    end = realtime();

#ifdef PROFILE
    cout << "Filter gradient gemm difference = " << end - begin << endl;
#endif


};

__global__ void inserthole(
    const float* grad,
    const int grad_height,
    const int grad_width,
    const int grad_channel,
    const int hole_grad_height,
    const int hole_grad_width,
    const int final_size,
    const int stride_row,
    const int stride_col,
    const int size,
    float* out
){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < final_size){
        const int batch_idx = blockIdx.y;
        grad += batch_idx * size;
        out += batch_idx * final_size;
        const int g_channel = index % grad_channel;
        const int g_channel_level_index = index / grad_channel;
        const int g_channel_level_col = g_channel_level_index % hole_grad_width;
        const int g_channel_level_row = g_channel_level_index / hole_grad_width;
        out+=index;
        float i_x = (hole_grad_width==grad_width)?g_channel_level_col:(g_channel_level_col/(float)stride_col);
        float i_y = (hole_grad_height==grad_height)?g_channel_level_row:(g_channel_level_row/(float)stride_row);
        const bool x = fmod(i_x,(float)1)==float(0);
        const bool y = fmod(i_y,(float)1)==float(0);
        *out = (x&y)?grad[g_channel+((int)i_x)*grad_channel+((int)i_y)*grad_channel*grad_width]:0;
        //*out = grad[g_channel+((int)i_x)*grad_channel+((int)i_y)*grad_channel*grad_width];
    }
}


void ConvamInputGradKernelLauncher(
    // grad needs pading and holes
    // im2col input
    const float* grad,
    float* holed_grad,
    float* im2col,
    const int real_grad_height,
    const int real_grad_width,
    const int hole_grad_width,
    const int hole_grad_height,
    const int back_pad_top,
    const int back_pad_left,
    const int back_pad_bottom,
    const int back_pad_right,
    const float* filter,
    //reverse and swap dimension 2 and 3 of the filters.s
    float* rsfilter,
    const int filter_height,
    const int filter_width,
    const int output_channel,
    const int stride_rows,
    const int stride_cols,
    // input related
    const int input_batch,
    const int input_height,
    const int input_width,
    const int input_channel,
    float* output
){
    double begin1 = realtime();
    if(hole_grad_height!=real_grad_height||hole_grad_width!=real_grad_width){
        // float holed[input_batch*hole_grad_width*hole_grad_height*output_channel];

        const int holed_size = hole_grad_width*hole_grad_height*output_channel;
        const int real_size = output_channel*real_grad_height*real_grad_width;
        dim3 dim_grid(ceil((float)holed_size/BLOCK_SIZE),input_batch);
        inserthole<<<dim_grid,BLOCK_SIZE>>>( grad, real_grad_height, real_grad_width, output_channel, hole_grad_height, hole_grad_width, holed_size,
            stride_rows, stride_cols, real_size, holed_grad);

            gpuErrchk( hipPeekAtLastError() );
            gpuErrchk( hipDeviceSynchronize() );
       im2colLauncher_Improved(
        holed_grad, input_batch, hole_grad_height, hole_grad_width, input_height, input_width,input_channel,output_channel,filter_height,
        filter_width,1,1,back_pad_left,back_pad_top,1,1,im2col);
   } else {
    im2colLauncher_Improved(
        grad, input_batch, hole_grad_height, hole_grad_width, input_height, input_width,input_channel,output_channel,filter_height,
        filter_width,1,1,back_pad_left,back_pad_top,1,1,im2col);
    }
    double end1 = realtime();

#ifdef PROFILE
    cout << "Error backpropagation: Im2Col time difference = " << end1 - begin1 << endl;
#endif


    const size_t m = input_batch*input_height*input_width; //4
    const size_t n = input_channel; //  1
    const size_t k = filter_width * filter_height * output_channel; //4
    const size_t lda = k; //4
    const size_t ldb = input_channel;
    const size_t ldc = input_channel;
    const int size = m*n;
    double begin =realtime();
    dim3 block_size(32,32);
    dim3 grid_size(ceil(filter_width * filter_height/(float)32.0), ceil(output_channel/(float)32.0));
    reverseNswapdim23<<<grid_size,block_size>>>(filter_height, filter_width, input_channel, output_channel, rsfilter, filter);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    dim3 blockSize(16, 16, 1);
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x, (m + blockSize.y - 1) / blockSize.y, 1);

    gemm<<<gridSize,blockSize,0>>>(m,n,k,im2col,lda,rsfilter,ldb,output,ldc);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    double end = realtime();
    // dim3 dim_grid1(ceil((float)size/BLOCK_SIZE),input_batch);
    // double begin = realtime();
    // gemm_inverse<<<dim_grid1,BLOCK_SIZE>>>(m,n,k,im2col,lda,filter,ldb,output,ldc,size,filter_width,filter_height,output_channel,input_channel);
    // gpuErrchk( hipPeekAtLastError() );
    // gpuErrchk( hipDeviceSynchronize() );
    // double end = realtime();

#ifdef PROFILE
    cout << "Error backpropagation: Gemm inverse time = " << end - begin << endl;
#endif

};
