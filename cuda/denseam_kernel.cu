#include "hip/hip_runtime.h"
#define EIGEN_USE_GPU

#include "gpu_kernel_helper.h"
#include "error.cuh"
#include "denseam.h"
#include "approx_mul_lut.h"

using namespace tensorflow;
using GpuDevice = Eigen::GpuDevice;

#ifdef FMBM32_MULTIPLIER
   #define MULTIPLY(a,b) FPmultMBM_fast32((a),(b));
   #include "FPmultMBM_fast32.inl"
#elif FMBM16_MULTIPLIER
    #define MULTIPLY(a,b) FPmultMBM_fast16((a),(b), lut, exp_lut);
    #include "FPmultMBM_fast16.inl"
#elif FMBM14_MULTIPLIER
    #define MULTIPLY(a,b) FPmultMBM_fast14((a),(b));
    #include "FPmultMBM_fast14.inl"
#elif FMBM12_MULTIPLIER
    #define MULTIPLY(a,b) FPmultMBM_fast12((a),(b));
    #include "FPmultMBM_fast12.inl"
#elif FMBM10_MULTIPLIER
    #define MULTIPLY(a,b) FPmultMBM_fast10((a),(b));
    #include "FPmultMBM_fast10.inl"
#elif MITCHEL32_MULTIPLIER
    #define MULTIPLY(a,b) FPmultMitch_fast32((a),(b));
    #include "Mitchell_32.inl"
#elif MITCHEL16_MULTIPLIER
    #define MULTIPLY(a,b) FPmultMitch_fast16((a),(b));
    #include "Mitchell_16.inl"
#elif MITCHEL14_MULTIPLIER
    #define MULTIPLY(a,b) FPmultMitch_fast14((a),(b));
    #include "Mitchell_14.inl"
#elif MITCHEL12_MULTIPLIER
    #define MULTIPLY(a,b) FPmultMitch_fast12((a),(b));
    #include "Mitchell_12.inl"
#elif MITCHEL10_MULTIPLIER
    #define MULTIPLY(a,b) FPmultMitch_fast10((a),(b));
    #include "Mitchell_10.inl"
#elif BFLOAT
    #define MULTIPLY(a,b) bfloat16mul((a),(b));
    #include "bfloat.inl"
#else
    #define MULTIPLY(a,b) ((a)*(b));
#endif

template <typename T>
__global__ void DenseamKernel(
    const T* inputs,
    const T* weights,
    const int batch, 
    const int units, 
    const int input_width, 
    T* output, 
    hipTextureObject_t lut,
    hipTextureObject_t exp_lut
    ) 
{ 
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x; 
    if(ix < units*batch)
    {
        int ix_unit = ix % units ;
        int ix_sample = ix / units;
        output[ix] = T(0);
        for (int ix_input = 0; ix_input < input_width; ix_input++)
        {
          output[ix] += MULTIPLY(inputs[ix_sample*input_width+ix_input], weights[ix_input*units+ix_unit]);
        }  
    }
};
template <typename T>
void DenseamFunctor<GpuDevice, T>::operator()(
        const GpuDevice& d, const T* inputs, const T* weights, T* output,
        const int batch, const int units, const int input_width,
        approx_mul_lut<GpuDevice>& mul_lut )
{ 
        unsigned blocksize = 1024;
        unsigned gridsize = (batch*units+blocksize -1)/blocksize;
        DenseamKernel<T><<<gridsize, blocksize, 0, d.stream()>>>(inputs, weights, batch, units, input_width, output, mul_lut.get_mant_mul_lut_text_(), mul_lut.get_exp_mul_lut_text_());
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
}

template <typename T>
__global__ void DenseamWeightsKernel(
    const T* grads,
    const T* inputs,
    const int input_width, 
    const int batch, 
    const int units, 
    T* grad_weights,
    hipTextureObject_t lut,
    hipTextureObject_t exp_lut
    ) 
{ 
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x; 
    if(ix < units*input_width)
    {
        int ix_unit = ix % units ;
        int ix_input = ix / units;
        grad_weights[ix] = T(0);
        for (int ix_sample = 0; ix_sample < batch; ix_sample++)
        {
           grad_weights[ix] += MULTIPLY(inputs[input_width*ix_sample+ix_input], grads[ix_sample*units+ix_unit]);
        }  
    }
};
template <typename T>
void DenseamWeightGradFunctor<GpuDevice, T>::operator()
    (const GpuDevice& d, const T* input, const T* grads,
            T* output, const int batch, const int units, const int input_width,
            approx_mul_lut<GpuDevice>& mul_lut ) 
            {
    unsigned blocksize = 1024;
    unsigned gridsize = (units*input_width+blocksize -1)/blocksize;
    DenseamWeightsKernel<T><<<gridsize, blocksize, 0, d.stream()>>>(grads, input, input_width, batch, units, output, mul_lut.get_mant_mul_lut_text_(), mul_lut.get_exp_mul_lut_text_());
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
}
template <typename T>
__global__ void DenseamInputKernel(
    const T* grads,
    const T* weights,
    const int input_width, 
    const int batch, 
    const int units, 
    T* grad_inputs, 
    hipTextureObject_t lut,
    hipTextureObject_t exp_lut
    ) 
{ 
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x; 
    if(ix < batch *input_width)
    {
        int ix_input = ix % input_width;
        int ix_sample = ix / input_width ;
        grad_inputs[ix] = T(0);

        for (int ix_unit = 0; ix_unit < units; ix_unit++)
        {
			grad_inputs[ix_sample*input_width+ix_input] += MULTIPLY(weights[ix_input*units+ ix_unit], grads[ix_sample*units+ix_unit]);
        }  
    }
};
template <typename T>
void DenseamInputGradFunctor<GpuDevice, T>::operator()
    (const GpuDevice& d, const T* weight, const T* grads,
            T* output, const int batch, const int units, const int input_width,
            approx_mul_lut<GpuDevice>& mul_lut
            ){
    unsigned blocksize = 1024;
    unsigned gridsize = (batch*input_width+blocksize -1)/blocksize;
    DenseamInputKernel<T><<<gridsize, blocksize, 0, d.stream()>>>(grads, weight, input_width, batch, units, output, mul_lut.get_mant_mul_lut_text_(), mul_lut.get_exp_mul_lut_text_());
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
}

template struct DenseamFunctor<GpuDevice, float>;
template struct DenseamFunctor<GpuDevice, int32>;
template struct DenseamInputGradFunctor<GpuDevice, float>;
template struct DenseamInputGradFunctor<GpuDevice, int32>;
template struct DenseamWeightGradFunctor<GpuDevice, float>;
template struct DenseamWeightGradFunctor<GpuDevice, int32>;
