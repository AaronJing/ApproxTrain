
#define EIGEN_USE_GPU

#include "gpu_kernel_helper.h"
#include "error.cuh"
#include "matmulam.h"
#include "approx_mul_lut.h"
#include "gemm.cuh"

using namespace tensorflow;
template <typename T>
void LaunchMatMul<Eigen::GpuDevice, T>::operator()(
      const Eigen::GpuDevice &d, const T* a, const T* b,
      const int batch_a, const int batch_b, const int row_a, const int col_a, const int row_b,
      const int col_b, T* out,
      approx_mul_lut<Eigen::GpuDevice>& mul_lut
      ){
    const uint32_t mant_mask = mul_lut.get_mant_mask_();
    const uint8_t a_shift = mul_lut.get_a_shift_();
    const uint8_t b_shift = mul_lut.get_b_shift_();
    const uint8_t mant_bitwidth = mul_lut.get_mant_width_();
    const int m = row_a;
    const int n = col_b;
    const int k = col_a;
    const int lda = col_a;
    const int ldb = col_b;
    const int ldc = col_b;
    dim3 blockSize(16, 16, 1);
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x, (m + blockSize.y - 1) / blockSize.y, 1);
    if (batch_a!=0&&batch_b!=0){
        if (batch_a > batch_b) {
            for(int i = 0; i < batch_a; i++){
                const T* temp_a = a + i*row_a*col_a;
                T* temp_c = out + i*row_a*col_b;
                gemm<T><<<gridSize, blockSize, 0, d.stream()>>>(m, n, k, 
                        temp_a, lda, b, ldb, temp_c, ldc, 
                        mul_lut.get_mant_mul_lut_text_(), 
                        mant_mask, a_shift, b_shift, mant_bitwidth);    
                gpuErrchk( hipPeekAtLastError() );
                gpuErrchk( hipDeviceSynchronize() );
            } 
        } else if (batch_a!=1 && batch_b!=1) {
            for(int i = 0; i < batch_a; i++){
                const T* temp_a = a + i*row_a*col_a;
                const T* temp_b = b + i*row_b*col_b;
                T* temp_c = out + i*row_a*col_b;
                gemm<T><<<gridSize, blockSize, 0, d.stream()>>>(m, n, k, 
                        temp_a, lda, temp_b, ldb, temp_c, ldc, 
                        mul_lut.get_mant_mul_lut_text_(), 
                        mant_mask, a_shift, b_shift, mant_bitwidth);    
                gpuErrchk( hipPeekAtLastError() );
                gpuErrchk( hipDeviceSynchronize() );
            } 
        } else {
            for(int i = 0; i < batch_a; i++){
                const T* temp_b = b + i*row_b*col_b;
                T* temp_c = out + i*row_a*col_b;
                gemm<T><<<gridSize, blockSize, 0, d.stream()>>>(m, n, k, 
                        a, lda, temp_b, ldb, temp_c, ldc, 
                        mul_lut.get_mant_mul_lut_text_(), 
                        mant_mask, a_shift, b_shift, mant_bitwidth);    
                gpuErrchk( hipPeekAtLastError() );
                gpuErrchk( hipDeviceSynchronize() );
            } 
        
        }
    } else if (batch_a!=0) {
        for(int i = 0; i < batch_a; i++){
            const T* temp_a = a + i*row_a*col_a;
            T* temp_c = out + i*row_a*col_b;
            gemm<T><<<gridSize, blockSize, 0, d.stream()>>>(m, n, k, 
                    temp_a, lda, b, ldb, temp_c, ldc, 
                    mul_lut.get_mant_mul_lut_text_(), 
                    mant_mask, a_shift, b_shift, mant_bitwidth);    
            gpuErrchk( hipPeekAtLastError() );
            gpuErrchk( hipDeviceSynchronize() );
        } 
    } else if (batch_b!=0) { 
        for(int i = 0; i < batch_b; i++){
            const T* temp_b = b + i*row_b*col_b;
            T* temp_c = out + i*row_a*col_b;
            gemm<T><<<gridSize, blockSize, 0, d.stream()>>>(m, n, k, 
                    a, lda, temp_b, ldb, temp_c, ldc, 
                    mul_lut.get_mant_mul_lut_text_(), 
                    mant_mask, a_shift, b_shift, mant_bitwidth);    
            gpuErrchk( hipPeekAtLastError() );
            gpuErrchk( hipDeviceSynchronize() );
        } 
    
    } else {
        gemm<T><<<gridSize, blockSize, 0, d.stream()>>>(m, n, k, 
                    a, lda, b, ldb, out, ldc, 
                    mul_lut.get_mant_mul_lut_text_(), 
                    mant_mask, a_shift, b_shift, mant_bitwidth);    
    
    }
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
}
template struct LaunchMatMul<Eigen::GpuDevice, float>;
template struct LaunchMatMul<Eigen::GpuDevice, int32>;
