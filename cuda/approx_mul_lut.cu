#define EIGEN_USE_GPU
#include <vector>
#include <fstream>
#include <iostream>
#include <algorithm>
#include <iterator>
#include "error.cuh"
#include "approx_mul_lut.h"
using namespace tensorflow;

template<>
class approx_mul_lut<Eigen::GpuDevice> : public approx_mul_lut_base {
    public:
        explicit approx_mul_lut(tensorflow::OpKernelConstruction* context);
        ~approx_mul_lut();
        auto get_mant_mul_lut_text_() -> hipTextureObject_t& {
            return mant_mul_lut_text_;
        }
        auto get_mant_mul_lut_() -> uint32_t* {
            return mant_mul_lut_cuda_;
        }
        auto get_exp_mul_lut_text_() -> hipTextureObject_t& {
            return exp_mul_lut_text_;
        }

};
approx_mul_lut<Eigen::GpuDevice>::approx_mul_lut(OpKernelConstruction * context):
            approx_mul_lut_base(context){

    gpuErrchk(hipMalloc(&mant_mul_lut_cuda_, 
            mant_mul_lut_.size() * sizeof(uint32_t)));
    gpuErrchk(hipMemcpy(mant_mul_lut_cuda_, mant_mul_lut_.data(),
            mant_mul_lut_.size()*sizeof(uint32_t), 
            hipMemcpyHostToDevice));
    hipResourceDesc mant_mul_lut_res_desc;
    memset(&mant_mul_lut_res_desc, 0, sizeof(hipResourceDesc));
    mant_mul_lut_res_desc.resType = hipResourceTypeLinear;
    mant_mul_lut_res_desc.res.linear.devPtr = mant_mul_lut_cuda_;
    mant_mul_lut_res_desc.res.linear.desc.f = 
        hipChannelFormatKindUnsigned;
    mant_mul_lut_res_desc.res.linear.desc.x = 32;
    mant_mul_lut_res_desc.res.linear.sizeInBytes = 
        mant_mul_lut_.size() * sizeof(uint32_t);
    
    hipTextureDesc mant_mul_text_desc;
    memset(&mant_mul_text_desc, 0, sizeof(hipTextureDesc));
    mant_mul_text_desc.readMode = hipReadModeElementType;
        
    gpuErrchk(hipCreateTextureObject(&mant_mul_lut_text_, &mant_mul_lut_res_desc, 
            &mant_mul_text_desc, nullptr));                

    gpuErrchk(hipMalloc(&exp_mul_lut_cuda_, 
            exp_mul_lut_.size() * sizeof(uint32_t)));
    gpuErrchk(hipMemcpy(exp_mul_lut_cuda_, exp_mul_lut_.data(),
            exp_mul_lut_.size()*sizeof(uint32_t), 
            hipMemcpyHostToDevice));
    hipResourceDesc exp_mul_lut_res_desc;
    memset(&exp_mul_lut_res_desc, 0, sizeof(hipResourceDesc));
    exp_mul_lut_res_desc.resType = hipResourceTypeLinear;
    exp_mul_lut_res_desc.res.linear.devPtr = exp_mul_lut_cuda_;
    exp_mul_lut_res_desc.res.linear.desc.f = 
        hipChannelFormatKindUnsigned;
    exp_mul_lut_res_desc.res.linear.desc.x = 32;
    exp_mul_lut_res_desc.res.linear.sizeInBytes = 
        exp_mul_lut_.size() * sizeof(uint32_t);
    
    hipTextureDesc exp_mul_text_desc;
    memset(&exp_mul_text_desc, 0, sizeof(hipTextureDesc));
    exp_mul_text_desc.readMode = hipReadModeElementType;
        
    gpuErrchk(hipCreateTextureObject(&exp_mul_lut_text_, &exp_mul_lut_res_desc, 
            &exp_mul_text_desc, nullptr));                
};

approx_mul_lut<Eigen::GpuDevice>::~approx_mul_lut(){
    hipDestroyTextureObject(mant_mul_lut_text_);
    hipFree(mant_mul_lut_cuda_);
    hipDestroyTextureObject(exp_mul_lut_text_);
    hipFree(exp_mul_lut_cuda_);
};
