#define EIGEN_USE_GPU
#include <vector>
#include <fstream>
#include <iostream>
#include <algorithm>
#include <iterator>
#include "error.cuh"
#include "approx_mul_lut.h"
using namespace tensorflow;

template<>
class approx_mul_lut<Eigen::GpuDevice> : public approx_mul_lut_base {
    public:
        explicit approx_mul_lut(tensorflow::OpKernelConstruction* context);
        ~approx_mul_lut();
        auto get_mant_mul_lut_text_() -> hipTextureObject_t& {
            return mant_mul_lut_text_;
        }
        auto get_mant_mul_lut_() -> uint8_t* {
            return mant_mul_lut_cuda_;
        }

};
approx_mul_lut<Eigen::GpuDevice>::approx_mul_lut(OpKernelConstruction * context):
            approx_mul_lut_base(context){
    gpuErrchk(hipMalloc(&mant_mul_lut_cuda_, 
            mant_mul_lut_.size() * sizeof(uint8_t)));
    gpuErrchk(hipMemcpy(mant_mul_lut_cuda_, mant_mul_lut_.data(),
            mant_mul_lut_.size()*sizeof(uint8_t), 
            hipMemcpyHostToDevice));
    hipResourceDesc mant_mul_lut_res_desc;
    memset(&mant_mul_lut_res_desc, 0, sizeof(hipResourceDesc));
    mant_mul_lut_res_desc.resType = hipResourceTypeLinear;
    mant_mul_lut_res_desc.res.linear.devPtr = mant_mul_lut_cuda_;
    mant_mul_lut_res_desc.res.linear.desc.f = 
        hipChannelFormatKindUnsigned;
    mant_mul_lut_res_desc.res.linear.desc.x = 8;
    mant_mul_lut_res_desc.res.linear.sizeInBytes = 
        mant_mul_lut_.size() * sizeof(uint8_t);
    
    hipTextureDesc mant_mul_text_desc;
    memset(&mant_mul_text_desc, 0, sizeof(hipTextureDesc));
    mant_mul_text_desc.readMode = hipReadModeElementType;
        
    gpuErrchk(hipCreateTextureObject(&mant_mul_lut_text_, &mant_mul_lut_res_desc, 
            &mant_mul_text_desc, nullptr));                

};

approx_mul_lut<Eigen::GpuDevice>::~approx_mul_lut(){
    hipDestroyTextureObject(mant_mul_lut_text_);
    hipFree(mant_mul_lut_cuda_);
};
